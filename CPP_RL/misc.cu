#include "hip/hip_runtime.h"
#include <array>
#include <complex>
#include <iostream>
#include <vector>
#include <complex>
//#include <thrust/device_vector.h>
#include "thrust/transforms.h"
#include "hipfft/hipfft.h"
#include "cufft_utils.h"

using namespace std;

/// Function to calculate the nearest 2^n that is greater than or equal to 2 * imgSize
__global__ void twoN(int *n, const int imgSize)
{
    /// Caculate nearest 2^n that is greater than or equal to 2 * imgSize
    *n = 1;
    while (*n < 2 * imgSize)
    {
        *n *= 2;
    }
}

/// Function to pad the matrix with zeros to the nearest power of 2 that is greater than or equal to 2 * imgSize
/// Assumes that the 2^n matrix is already allocated (Global functions can't allocate memory nor return values)
__global__ void padMatrix(const thrust::device_vector<thrust::float> *input, thrust::device_vector<thrust::complex> *output, 
const int *imgSize_x, const int *imgSize_y, const int *n, const int *m)
{
    // Pads matrix with zeros to the nearest power of 2 that is greater than or equal to 2 * imgSize
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;

    if (idx < *n && idy < *m)
    {
        if (idx < *imgSize_x && idy < *imgSize_y)
        {
            output[idx * (*m) + idy] = input[idx * (*imgSize_y) + idy];
        }
        else
        {
            output[idx * (*m) + idy] = 0.0f;
        }
    }
}

/// Function to reset the padding to zero if for some reason the padding is not zero
__global__ void setPadZero(thrust::device_vector<thrust::complex> *input, const int *n, const int *m)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;

    if (idx >= *n || idy >= *m)
    {
        input[idx * (*m) + idy] = 0.0f;
    }
}

// Function to drop the imaginary part of the complex matrix
__global__ void dropImag(thrust::device_vector<thrust::complex> *input)
{
    input.imag(0.0f);
}


// __host__ void kernfft(const float *kern, thrust::device_vector<thrust::complex> *result, const int n, const int m)
// {
//     /// Take the 2D FFT
//     hipfftHandle plan;
//     hipfftPlan2d(&plan, n, m, HIPFFT_Z2Z);
//     hipfftExecZ2Z(plan, kern, result, HIPFFT_FORWARD);
//     hipfftDestroy(plan);
// }

// __host__ void fftconv(int batch, int dims, const float *img, const float *kernfft, float *result)
// {
//     /// Take the 2D FFT
//     hipfftHandle plan;
//     hipfftPlanMany(&plan, 2, dims, NULL, 1, 0, NULL, 1, 0, HIPFFT_C2C, batch);
//     hipfftExecZ2Z(plan, img, img, HIPFFT_FORWARD);
//     hipfftDestroy(plan);

//     /// Thurst element wise multiplication (treating array like vectors)
//     thrust::transform(img.begin(), img.end(), kernfft.begin(), result.begin(), thrust::multiplies<cudaDoubleComplex>());

//     /// Take the inverse 2D FFT
//     hipfftPlan2d(&plan, n, m, HIPFFT_Z2Z);
//     hipfftExecC2C(plan, result, result, HIPFFT_BACKWARD);
//     hipfftDestroy(plan);

//     /// Drop the imaginary part of the complex matrix and crop the result to the original size
//     dropImag(result);
// }

__global__ void ogCrop(const thrust::device_vector<thrust::complex> *input, thrust::host_vector<thrust::float> *output, 
const int n, const int m, const int imgSize_x, const int imgSize_y, cosnt int numKern)
{
    /// Input is a complex matrix of n x m x numKern
    /// Output is a real matrix of imgSize_x x imgSize_y x numKern
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;
    int idz = blockIdx.z * blockDim.z + threadIdx.z;

    if (idx < imgSize_x && idy < imgSize_y && idz < numKern)
    {
        output[idx * imgSize_y + idy + idz * imgSize_x * imgSize_y] = static_cast<float>(input[idx * m + idy + idz * n * m].real()); // copy-assingment
    }
}

__global__ void fftNorm(thrust::device_vector<thrust::complex> *input, const int n, const int m, const int numKern)
{
    /// Normalize the FFT of the kernel
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;
    int idz = blockIdx.z * blockDim.z + threadIdx.z;

    if (idx < n && idy < m && idz < numKern)
    {
        input[idx * m + idy + idz * n * m] /= static_cast<float>(n * m);
    }
}

__global__ void fftbNorm(thrust::device_vector<thrust::complex> *input, const int n, const int m, const int numKern)
{
    /// Normalize the FFT of the back kernel
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;
    int idz = blockIdx.z * blockDim.z + threadIdx.z;

    if (idx < n && idy < m && idz < numKern)
    {
        input[idx * m + idy + idz * n * m] /= static_cast<float>(n * m);
    }
}
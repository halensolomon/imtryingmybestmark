#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <hipDNN.h>
#include <algorithm>
#include <opencv2/imgcodecs.hpp>
#include "cuda_rl.cuh"
#include "algorithms.cuh"
#include "fftconv.cuh"
#include "file_io.cuh"

namespace fs = std:filesystem;
typedef float2 Complex;

int main(int argc, char** argv)
{
    hipError_t error;

    int numGPUs;
    hipGetDeviceCount(&numGPUs);
    std::cout << "Number of GPUs: " << numGPUs << std::endl;

        // Use all available GPUs
    for (int i = 0; i < numGPUs; i++)
    {
        hipSetDevice(i);
        hipDeviceEnablePeerAccess(i, 0);
    }

    // Search for all the images in the directory
    std::vector<std::string> imgPaths;
    std::vector<std::string> kernPaths;
    std::string imgPath = "C:/some/path/to/images/";
    std::string kernPath = "C:/some/path/to/kernels/";

    imgPaths = fileSearch(imgPath, ".tif");
    int numImages = imgPaths.size();

    kernPaths = fileSearch(kernPath, ".tif");
    int numKernels = kernPaths.size();
    
    // Find the size of the image and kernel
    ImageData testimg = readImage(imgPaths[0]);
    ImageData testkern = readImage(kernPaths[0]);

    imgx = testimg.width;
    imgy = testimg.height;
    kernx = testkern.width;
    kerny = testkern.height;

    // Allocate memory for the image and kernel data
    float* imgdevptr;
    float* kerndevptr;
    float* result2ddevptr;
    float* result3ddevptr;

    hipError_t imgmem, kernmem, resultmem;

    imgmem = hipMalloc((void**)&imgdevptr, imgx * imgy * sizeof(float)); // Should only store one image at a time for memory efficiency
    kernmem = hipMalloc((void**)&kerndevptr, kernx * kerny * numKernels * sizeof(float)); // NEED to store all the kernels at once
    resultmem = hipMalloc((void**)&result2ddevptr, imgx * imgy * sizeof(float)); // Should only store one image at a time for memory efficiency
    resultmem = hipMalloc((void**)&result3ddevptr, imgx * imgy * numKernels * sizeof(float)); // NEED to store all the results at once

    // Copy kernel data to device
    for (int i = 0; i < numKernels; i++)
    {
        ImageData kerndata = readImage(kernPaths[i]);
        hipMemcpy(kerndevptr + i * kernx * kerny, kerndata.data, kernx * kerny * sizeof(float), hipMemcpyHostToDevice); 
    }

    if (kernmem != hipSuccess)
    {
        std::cerr << "Failed to allocate memory for the kernel on the device" << std::endl;
        exit(1);
    }

    float* backkerndevptr;

    hipError_t backkernmem;

    backkernmem = hipMalloc((void**)&backkerndevptr, kernx * kerny * numKernels * sizeof(float)); // NEED to store all the kernels at once

    // Copy kernel data to device, but backwards
    for (int i = 0; i < numKernels; i++)
    {
        ImageData kerndata = readImage(kernPaths[i]);
        std::reverse(kerndata.data.begin(), kerndata.data.end());
        hipMemcpy(backkerndevptr + i * kernx * kerny, kerndata.data, kernx * kerny * sizeof(float), hipMemcpyHostToDevice);
    }

    if (backkernmem != hipSuccess)
    {
        std::cerr << "Failed to allocate memory for the kernel on the device" << std::endl;
        exit(1);
    }

    thrust::device_vector<float> kernsum(kernx * kerny); // Will be used to normalize the kernel

    // Normalize the forward kernel
    for (int i = 0; i < numKernels; i++)
    {
        float forward_sum = thrust::reduce(kerndevptr + i * kernx * kerny, kerndevptr + (i + 1) * kernx * kerny);
        forward_sum += 1e-6; // Add a small number to avoid division by zero
        thrust::device_vector<float> kernvec = thurst::device_vector<float>(kerndevptr + i * kernx * kerny, kerndevptr + (i + 1) * kernx * kerny);
        thurst::transform(kernvec.begin(), kernvec.end(), thrust::make_constant_iterator(forward_sum), kernvec.begin(), thurst::divide<float>());

        thrust::transform(backkernvec.begin(), backkernvec.end(), kernsum.begin(), kernsum.begin(), thurst::add<float>());

        // Copy the kernel back to the device
        hipMemcpy(kerndevptr + i * kernx * kerny, kernvec.data(), kernx * kerny * sizeof(float), hipMemcpyHostToDevice);
        
        // Don't forget to clear the vector
        kernvec.clear();
        kernvec.shrink_to_fit();
    }

    // Normalize the kernels
    thrust::transform(kernsum.begin(), kernsum.end(), thrust::make_constant_iterator(numKernels), kernsum.begin(), thurst::multiply<float>());
    thrust::transform(kernsum.begin(), kernsum.end(), thrust::make_constant_iterator(1e-6), kernsum.begin(), thurst::add<float>());

    // Make the backward kernel
    for (int i = 0; i < numKernels; i++)
    {
        thrust::device_vector<float> backkern = thurst::device_vector<float>(backkerndevptr + i * kernx * kerny, backkerndevptr + (i + 1) * kernx * kerny);
        thrust::transform(backkern.begin(), backkern.end(), kernsum.begin(), backkern.begin(), thurst::divide<float>());
        hipMemcpy(backkerndevptr + i * kernx * kerny, backkern.data(), kernx * kerny * sizeof(float), hipMemcpyHostToDevice);

        backkern.clear();
        backkern.shrink_to_fit();
    }
    // Clear the kernel sum
    kernsum.clear();
    kernsum.shrink_to_fit();

    /// Read images sequentially
    for (i = 0, i < Images, i++)
    {
        std::vector<float>* imgPtr = readImage(imgPaths[i]);
        if (imgPtr != nullptr)
        {
            std::cout << "Image read successfully" << std::endl;
            float* imgPinnedMem; // Pinned memory for image data
            size_t imgByteSize = imgPtr->size() * sizeof(float); // Size of the image in bytes
            hipError_t error = hipHostAlloc((void**)&imgPinnedMem, imgByteSize); // Allocate pinned memory for image data

            // Copy data from the vector to pinned memory
            hipMemcpy(imgPinnedMem, imgPtr->data(), imgByteSize); // Copy the image data to pinned memory

            if (error != hipSuccess) 
            {
                std::cerr << "Failed to allocate pinned memory for image data" << std::endl;
                //delete ImagePtr; // Release ImagePtr if allocation fails
            }

        std::cout << "Image size is: " << imgPtr->size() << std::endl;
        std::cout << "Kernel size is: " << kernPtr->size() << std::endl;

        if (imgPtr->size() != kernPtr->size())
            {
            std::cerr << "Image and kernel sizes do not match" << std::endl;
            exit(2);
            }
        }

        // Continue with the algorithm
        rlAlg(imgdevptr, kerndevptr, backkerndevptr, result2ddevptr, result3ddevptr, imgPtr->size(), kernPtr->size(), numKernels, 400);
    }
}